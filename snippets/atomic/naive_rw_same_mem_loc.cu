#include "hip/hip_runtime.h"
// Compile command: nvcc naive_rw_same_mem_loc.cu -o naive_rw_same_mem_loc

#include <stdio.h>
#include "gputimer.h"

#define NUM_THREADS 1000000
#define BLOCK_WIDTH 1000

#define ARRAY_SIZE  100



// Helper function to print an array
void print_array(int *array, int size)
{
    printf("{ ");
    for (int i = 0; i < size; i++) { 
        printf("%d ", array[i]); 
    }
    printf("}\n");
}

// Naive implementation returns WRONG result!
__global__ void increment_naive(int *g)
{
    // Mapping from block grid to thread identity
    int i = blockIdx.x * blockDim.x + threadIdx.x; 

    // Each thread to increment consecutive elements, wrapping at ARRAY_SIZE
    i = i % ARRAY_SIZE;  
    g[i] = g[i] + 1;  // "read-modify-write" operation causes problem if multiple threads 
                      // access same memory location at the same time
}

int main(int argc,char **argv)
{   
    GpuTimer timer;
    printf("%d total threads in %d blocks writing into %d array elements\n",
           NUM_THREADS, NUM_THREADS / BLOCK_WIDTH, ARRAY_SIZE);

    // Declare and allocate host memory
    int h_array[ARRAY_SIZE];
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);
 
    // Declare, allocate, and zero out GPU memory
    int * d_array;
    hipMalloc((void **) &d_array, ARRAY_BYTES);
    hipMemset((void *) d_array, 0, ARRAY_BYTES); 

    // Launch the kernel 
    timer.Start();
    increment_naive<<<NUM_THREADS/BLOCK_WIDTH, BLOCK_WIDTH>>>(d_array);
    timer.Stop();
    
    // Copy the result from device to the host
    hipMemcpy(h_array, d_array, ARRAY_BYTES, hipMemcpyDeviceToHost);

    // Print resulting array and elapsed time
    print_array(h_array, ARRAY_SIZE);
    printf("Time elapsed = %g ms\n", timer.Elapsed());
 
    // Free GPU memory allocation and exit
    hipFree(d_array);

    return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

#define N 64
#define BLOCK_DIM 32

__global__ void matrixMultiply(int *d_a, int *d_b, int *d_out, int width){
	
    // Mapping from 2D block grid to absolute 2D locations on C matrix
    int idx_x = blockDim.x * blockIdx.x + threadIdx.x;
    int idx_y = blockDim.y * blockIdx.y + threadIdx.y;
  
	// Multiply
	int sum = 0;
	if (idx_x < width && idx_y < width){
		for (int k = 0; k < width; k++){
			sum += d_a[idx_x * width + k] * d_b[k * width + idx_y];
		}
		
	    // 2D location in C matrix to global memory 1D offset
	    int index = idx_x * width + idx_y;  // Row-major order with 0 based indices
		d_out[index] = sum;
	}
}

int main(){
	// Declare 2D matrices on host
	int h_a[N][N], h_b[N][N], h_out[N][N];
	
	// Declare device/GPU memory pointers
	int *d_a, *d_b, *d_out;
	
	// Memory size
	int size = N * N * sizeof(int);
	
	// Initialize matrices on host
	for (int i=0; i<N; i++){
		for (int j=0; j<N; j++){
			h_a[i][j] = 1;  // Matrix A
			h_b[i][j] = 2;  // Matrix B
		}
	}
	
	// Allocate GPU memory
	hipMalloc((void **) &d_a, size);
	hipMalloc((void **) &d_b, size);
	hipMalloc((void **) &d_out, size);
	
	// Transfer input matrices from host to device
	hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
	
	// Define grid blocks dimensions
	dim3 blockSize(BLOCK_DIM, BLOCK_DIM);
	dim3 gridSize((int)ceil(N/blockSize.x), (int)ceil(N/blockSize.y));
	
	// Launch the kernel
	matrixMultiply<<<gridSize, blockSize>>>(d_a, d_b, d_out, N);
	
	// Copy the result from device to the host
	hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);
	
	// Print out the sum of output matrix elements
	int total = 0;
	for (int i=0; i<N; i++){
		for (int j=0; j<N; j++){
			total += h_out[i][j];
		}
	}
	std::cout << "Total: " << total << std::endl;
	
	// Free GPU memory allocation
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_out);
	
	return 0;
}

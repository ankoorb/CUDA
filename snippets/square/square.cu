
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void square(float* d_out, float* d_in){
	int idx = threadIdx.x;
	float f = d_in[idx];
	d_out[idx] = f * f;
}

int main(int argc, char ** argv){
	const int ARRAY_SIZE = 64;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

	// Generate input array on the host
	float h_in[ARRAY_SIZE];
	for (int i = 0; i < ARRAY_SIZE; i++){
		h_in[i] = float(i);
	}

	float h_out[ARRAY_SIZE];

	// Declare GPU memory pointers
	float* d_in;
	float* d_out;

	// Allocate GPU memory
	hipMalloc((void **) &d_in, ARRAY_BYTES); 
	hipMalloc((void **) &d_out, ARRAY_BYTES); 

	/*
	 * cudaMalloc() needs to modify the given pointer (the pointer itself 
	 * not what the pointer points to), so you need to pass "void**" which 
	 * is a pointer to the pointer.
	 */

	 // Transfer the array to the GPU
	 hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

	 // Launch the kernel
	 square<<<1, ARRAY_SIZE>>>(d_out, d_in);

	 // Copy back the result array to the CPU
	 hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	 // Print out the resulting array
	 for (int i = 0; i < ARRAY_SIZE; i++){
	 	printf("%f", h_out[i]);
	 	printf(((i % 4) != 3) ? "\t" : "\n");
	 }

	 // Free GPU memory allocation
	 hipFree(d_in);
	 hipFree(d_out);

	 return 0;

}


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

#define N 64
#define BLOCK_DIM 32

__global__ void matrixTranspose(int *d_a, int *d_out, int width){
	
    // Mapping from 2D block grid to absolute 2D locations on C matrix
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    
	// Transpose
	if (row < width && col < width){		
		d_out[row * width + col] = d_a[col * width + row];  
	}
}

int main(){
	// Declare 2D matrices on host
	int h_a[N][N], h_out[N][N];
	
	// Declare device/GPU memory pointers
	int *d_a, *d_out;
	
	// Memory size
	int size = N * N * sizeof(int);
	
	// Initialize matrices on host
	for (int i=0; i<N; i++){
		for (int j=0; j<N; j++){
			if (i % 2 == 0){
				h_a[i][j] = i + j;  // Matrix A
			}
			else {
				h_a[i][j] = i * j;  // Matrix A
			}
			
		}
	}
	
	// Allocate GPU memory
	hipMalloc((void **) &d_a, size);
	hipMalloc((void **) &d_out, size);
	
	// Transfer input matrices from host to device
	hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
	
	// Define grid blocks dimensions
	dim3 blockSize(BLOCK_DIM, BLOCK_DIM);
	dim3 gridSize((int)ceil(N/blockSize.x), (int)ceil(N/blockSize.y));
	
	// Launch the kernel
	matrixTranspose<<<gridSize, blockSize>>>(d_a, d_out, N);
	
	// Copy the result from device to the host
	hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

	// Print out the transposed array
	for (int i=0; i<N; i++){
		for (int j=0; j<N; j++){
			printf("A[%d,%d]: %d\t", i, j, h_a[i][j]);
			printf("AT[%d,%d]: %d", i, j, h_out[i][j]);
			printf("\n");
		}
	}
	
	// Free GPU memory allocation
	hipFree(d_a);
	hipFree(d_out);
	
	return 0;
}
